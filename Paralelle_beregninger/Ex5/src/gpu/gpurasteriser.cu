#include "hip/hip_runtime.h"
#include "gpurasteriser.cuh"
#include "utilities/OBJLoader.hpp"
#include <vector>
#include <iomanip>
#include <chrono>
#include <limits>
#include <iostream>
#include <algorithm>
#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include "utilities/cuda_error_helper.hpp"
#include <chrono>

// Todo: Implement

////////////////////////////////////////////
// New code

class NEW_timer_t{
public:
	std::chrono::high_resolution_clock::time_point startTime;

	void start(){
		this-> startTime = std::chrono::high_resolution_clock::now();
	}
	void stop(){
		std::cout <<
				 std::chrono::duration_cast<std::chrono::milliseconds>( (std::chrono::high_resolution_clock::now() - startTime)).count()
		<<"\n";
		}
};
NEW_timer_t timer;
////////////////////////////////////////////


// UTILITY FUNCTIONS HAVE BEEN MOVED INTO THE KERNEL SOURCE FILE ITSELF
// CUDA relocatable and separable compilation is possible, but due to the many possible
// problems it can cause on different platforms, I decided to take the safe route instead
// and make sure it would compile fine for everyone. That implies moving everything into
// one file unfortunately.

class globalLight {
public:
	float3 direction;
	float3 colour;
	__host__ __device__ globalLight(float3 const vdirection, float3 const vcolour) : direction(vdirection), colour(vcolour) {}
};

__host__ __device__ float dotGPU(float3 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ float3 normalizeGPU(float3 v)
{
    float invLen = 1.0f / sqrtf(dotGPU(v, v));
    v.x *= invLen;
    v.y *= invLen;
    v.z *= invLen;
    return v;
}

// Utility function if you'd like to convert the depth buffer to an integer format.
__host__ __device__ int depthFloatToInt(float value) {
	value = (value + 1.0f) * 0.5f;
    return static_cast<int>(static_cast<double>(value) * static_cast<double>(16777216));
}

__host__ __device__ bool isPointInTriangle(
		float4 const &v0, float4 const &v1, float4 const &v2,
		unsigned int const x, unsigned int const y,
		float &u, float &v, float &w) {
		u = (((v1.y - v2.y) * (x    - v2.x)) + ((v2.x - v1.x) * (y    - v2.y))) /
				 	 (((v1.y - v2.y) * (v0.x - v2.x)) + ((v2.x - v1.x) * (v0.y - v2.y)));
		if (u < 0) {
			return false;
		}
		v = (((v2.y - v0.y) * (x    - v2.x)) + ((v0.x - v2.x) * (y    - v2.y))) /
					(((v1.y - v2.y) * (v0.x - v2.x)) + ((v2.x - v1.x) * (v0.y - v2.y)));
		if (v < 0) {
			return false;
		}
		w = 1 - u - v;
		if (w < 0) {
			return false;
		}
		return true;
}

__host__ __device__ float3 computeInterpolatedNormal(
		float3 const &normal0,
		float3 const &normal1,
		float3 const &normal2,
		float3 const &weights
	) {
	float3 weightedN0, weightedN1, weightedN2;

	weightedN0.x = (normal0.x * weights.x);
	weightedN0.y = (normal0.y * weights.x);
	weightedN0.z = (normal0.z * weights.x);

	weightedN1.x = (normal1.x * weights.y);
	weightedN1.y = (normal1.y * weights.y);
	weightedN1.z = (normal1.z * weights.y);

	weightedN2.x = (normal2.x * weights.z);
	weightedN2.y = (normal2.y * weights.z);
	weightedN2.z = (normal2.z * weights.z);

	float3 weightedNormal;

	weightedNormal.x = weightedN0.x + weightedN1.x + weightedN2.x;
	weightedNormal.y = weightedN0.y + weightedN1.y + weightedN2.y;
	weightedNormal.z = weightedN0.z + weightedN1.z + weightedN2.z;

	return normalizeGPU(weightedNormal);
}

__host__ __device__ float computeDepth(
		float4 const &v0, float4 const &v1, float4 const &v2,
		float3 const &weights) {
	return weights.x * v0.z + weights.y * v1.z + weights.z * v2.z;
}





// ORIGINAL SOURCE FILE IS STARTING HERE

struct workItemGPU {
    float scale;
    float3 distanceOffset;

    workItemGPU(float& scale_, float3& distanceOffset_) : scale(scale_), distanceOffset(distanceOffset_) {}
    __host__ __device__ workItemGPU() : scale(1), distanceOffset(make_float3(0, 0, 0)) {}
};

__device__ void runVertexShader( float4 &vertex,
                      float3 positionOffset,
                      float scale,
					  unsigned int const width,
					  unsigned int const height,
				  	  float const rotationAngle = 0)
{
	float const pi = 3.1415926f;
	// The matrices defined below are the ones used to transform the vertices and normals.

	// This projection matrix assumes a 16:9 aspect ratio, and an field of view (FOV) of 90 degrees.
	mat4x4 const projectionMatrix(
		0.347270,   0, 			0, 		0,
		0,	  		0.617370, 	0,		0,
		0,	  		0,			-1, 	-0.2f,
		0,	  		0,			-1,		0);

	mat4x4 translationMatrix(
		1,			0,			0,			0 + positionOffset.x /*X*/,
		0,			1,			0,			0 + positionOffset.y /*Y*/,
		0,			0,			1,			-10 + positionOffset.z /*Z*/,
		0,			0,			0,			1);

	scale *= 3.0f;
	mat4x4 scaleMatrix(
		scale/*X*/,	0,			0,				0,
		0, 			scale/*Y*/, 0,				0,
		0, 			0,			scale/*Z*/, 	0,
		0, 			0,			0,				1);

	mat4x4 const rotationMatrixX(
		1,			0,				0, 				0,
		0, 			cosf(0), 	-sinf(0),	0,
		0, 			sinf(0),	cosf(0), 	0,
		0, 			0,				0,				1);

	float const rotationAngleRad = (pi / 4.0f) + (rotationAngle / (180.0f/pi));

	mat4x4 const rotationMatrixY(
		cosf(rotationAngleRad), 0, sinf(rotationAngleRad), 0,
		0, 1, 0, 0,
		-sinf(rotationAngleRad), 0, cosf(rotationAngleRad), 	0,
		0, 0, 0, 1);

	mat4x4 const rotationMatrixZ(
		cosf(pi),	-sinf(pi),	0,			0,
		sinf(pi), 	cosf(pi), 	0,			0,
		0,				0,				1,			0,
		0, 				0,				0,			1);

	mat4x4 const MVP =
		projectionMatrix * translationMatrix * rotationMatrixX * rotationMatrixY * rotationMatrixZ * scaleMatrix;

		float4 transformed = (MVP * vertex);

    vertex.x = transformed.x / transformed.w;
    vertex.y = transformed.y / transformed.w;
    vertex.z = transformed.z / transformed.w;
    vertex.w = 1.0;

    vertex.x = (vertex.x + 0.5f) * (float) width;
    vertex.y = (vertex.y + 0.5f) * (float) height;
}


__device__ float3 runFragmentShader(
						GPUMesh &mesh,
						unsigned int triangleIndex,
						float3 const &weights)
{
	float3 normal = computeInterpolatedNormal(
            mesh.normals[3 * triangleIndex + 0],
            mesh.normals[3 * triangleIndex + 1],
            mesh.normals[3 * triangleIndex + 2],
			weights);

    float3 colour = make_float3(0.0f, 0.0f, 0.0f);

    const unsigned int lightSourceCount = 1;
    const globalLight lightSources[lightSourceCount] = {{make_float3(0.3f, 0.5f, 1.0f), make_float3(1.0f, 1.0f, 1.0f)}};

	for (int lightSource = 0; lightSource < lightSourceCount; lightSource++) {
		globalLight l = lightSources[lightSource];
		float lightNormalDotProduct =
			normal.x * l.direction.x + normal.y * l.direction.y + normal.z * l.direction.z;

		float3 diffuseReflectionColour;
		diffuseReflectionColour.x = mesh.objectDiffuseColour.x * l.colour.x;
		diffuseReflectionColour.y = mesh.objectDiffuseColour.y * l.colour.y;
		diffuseReflectionColour.z = mesh.objectDiffuseColour.z * l.colour.z;

		colour.x += diffuseReflectionColour.x * lightNormalDotProduct;
		colour.y += diffuseReflectionColour.y * lightNormalDotProduct;
		colour.z += diffuseReflectionColour.z * lightNormalDotProduct;
	}

    colour.x = fminf(fmaxf(colour.x, 0.0f), 1.0f);
    colour.y = fminf(fmaxf(colour.y, 0.0f), 1.0f);
    colour.z = fminf(fmaxf(colour.z, 0.0f), 1.0f);

    return colour;
}







////////////////////////////////////////////////
// New code

// We iterate over each pixel in the triangle's bounding box
__device__ void iterateOverTriangle(	int minx,
													int miny,
													int maxx,
													int maxy,
													GPUMesh &mesh,
													unsigned int triangleIndex,
													float4 v0,
													float4 v1,
													float4 v2,
													unsigned char* frameBuffer,
													int* depthBuffer,
													int width
	){
	for (unsigned int x = minx; x < maxx; x++) {
		for (unsigned int y = miny; y < maxy; y++) {
			float u, v, w;
			// For each point in the bounding box, determine whether that point lies inside the triangle
			if (isPointInTriangle(v0, v1, v2, x, y, u, v, w)) {
				// If it does, compute the distance between that point on the triangle and the screen
				float pixelDepth = computeDepth(v0, v1, v2, make_float3(u, v, w));
				// If the point is closer than any point we have seen thus far, render it.
				// Otherwise it is hidden behind another object, and we can throw it away
				// Because it will be invisible anyway.
								if (pixelDepth >= -1 && pixelDepth <= 1)
								{
									int myDepth = depthFloatToInt(pixelDepth);
					int newDepth = atomicMin(&depthBuffer[y * width + x], myDepth);

					// I realise this does not solve the race condition.
										// However, it does reduce the probability it occurs.
										// Solving this properly requires implementing a full-blown tile renderer.
										// And I think it's more important to keep things as simple as possible here,
										// so you can understand what is going on.
					if(myDepth < newDepth) {
						float3 pixelColour = runFragmentShader(mesh, triangleIndex, make_float3(u, v, w));

						if(myDepth == depthBuffer[y * width + x]) {
								frameBuffer[4 * (x + (width * y)) + 0] = pixelColour.x * 255.0f;
								frameBuffer[4 * (x + (width * y)) + 1] = pixelColour.y * 255.0f;
								frameBuffer[4 * (x + (width * y)) + 2] = pixelColour.z * 255.0f;
								frameBuffer[4 * (x + (width * y)) + 3] = 255;
						}
					}
				}
			}
		}
	}
}

__device__ float4 sub(const float4& v0, const float4& v1){
			float4 out;
			out.x = v0.x-v1.x;
			out.y = v0.y-v1.y;
			out.z = v0.z-v1.z;
			out.w = v0.w-v1.w;
			return out;


}
__device__ float sq(float in){
	return in*in;
}
__device__ float chross(const float4& v0, const float4& v1){
		float out=0;
		out+=sq(v0.x*v1.y - v1.x*v0.y); //z²
		out+=sq(v1.x*v0.z - v0.x*v1.z); //z²
		out+=sq(v0.y*v1.z - v1.z*v0.y); //z²
		return out;
}
__device__ bool f_isLarge(const float4& v0, const float4& v1, const float4& v2){
			float4 edges[3];
			edges[0] = sub(v1,v0);
			edges[1] = sub(v2,v0);
			float area= chross(edges[0], edges[1]);

			return area > 10000;
}

__device__ float4 getVec(const float4 &base, int source){
				float4 out;
				out.x = __shfl_sync(0xFFFFFFFF, base.x, source);
				out.y = __shfl_sync(0xFFFFFFFF, base.y, source);
				out.z = __shfl_sync(0xFFFFFFFF, base.z, source);
				out.w = __shfl_sync(0xFFFFFFFF, base.w, source);

				// w is always 1
				return out;

}

////////////////////////////////////////////////





/**
 * The main procedure which rasterises all triangles on the framebuffer
 * @param transformedMesh         Transformed mesh object
 * @param frameBuffer             frame buffer for the rendered image
 * @param depthBuffer             depth buffer for every pixel on the image
 * @param width                   width of the image
 * @param height                  height of the image
 */
__device__ void rasteriseTriangle( float4 &v0, float4 &v1, float4 &v2,
                        GPUMesh &mesh,
                        unsigned int triangleIndex,
                        unsigned char* frameBuffer,
                        int* depthBuffer,
                        unsigned int const width,
                        unsigned int const height ) {

    // Compute the bounding box of the triangle.
    // Pixels that are intersecting with the triangle can only lie in this rectangle

	int minx = int(floorf(fminf(fminf(v0.x, v1.x), v2.x)));
	int maxx = int(ceilf(fmaxf(fmaxf(v0.x, v1.x), v2.x)));
	int miny = int(floorf(fminf(fminf(v0.y, v1.y), v2.y)));
	int maxy = int(ceilf(fmaxf(fmaxf(v0.y, v1.y), v2.y)));

	// Make sure the screen coordinates stay inside the window
    // This ensures parts of the triangle that are outside the
    // view of the camera are not drawn.
	minx = max(minx, (unsigned int) 0);
	maxx = min(maxx, width);
	miny = max(miny, (unsigned int) 0);
	maxy = min(maxy, height);

	////////////////////////////////////////////////
	// New code

	// For some reason, dividing the work into a grid is disasterous, while lines work fine.
	// So we do that to be safe


	// Debugging is quite hart here, since printf causes a silent crash.
	// It might have something to do qith limited register space or cache space(?)
	int blockSize = (maxx -minx)*(maxy-miny);
	bool isLarge = f_isLarge(v0,v1,v2);
	if(true &&isLarge){
		int x =0;
		x =   threadIdx.x ;


		// x += ((threadIdx.y-1)); // treadIdx-y has started to be in the range [8..15]
		// printf("g(%i, %i, %i), %i, (%i, %i)\n",
		// 				blockIdx.x, blockIdx.y, blockIdx.z,
		// 				triangleIndex,
		// 				x, blockDim.x
		// 			);


		// int y = threadIdx.x/8;
		// There were some overlap-issues, so we go for an overkill
		int Nminx = ( x   *(maxx - minx))/32  + minx;
		int Nmaxx = ((x+1)*(maxx - minx))/32 + minx;
		// int Nminy =  (    y * (maxy-miny))/4 + miny;
		// int Nmaxy =  ((1+ y)* (maxy-miny))/4 + miny;

		// miny = Nminy;
		// maxy = Nmaxy;
		maxx = x == 31 ? maxx : Nmaxx;
		minx = x ==0 ? minx : Nminx;

		// Forces the check to be inside the frame
	minx = max(minx, (unsigned int) 0);
	maxx = min(maxx, width);
	miny = max(miny, (unsigned int) 0);
	maxy = min(maxy, height);

	}




	iterateOverTriangle(			minx,
														miny,
														maxx,
														maxy,
														mesh,
														triangleIndex,
														v0,
														v1,
														v2,
														frameBuffer,
														depthBuffer,
														width);


	////////////////////////////////////////////////

// Removed the loop and made it a function

}





////////////////////////////////////////////////////////////////////
// New code

class NEW_GPUQueue{
	// Work queue on the GPU
public:

	float* scales;
	float* xList;
	float* yList;
	float* zList;
	unsigned long length;

   __device__  inline workItemGPU get(const int index) const{

		workItemGPU output;
		output.scale = scales[index];
		output.distanceOffset.x = xList[index];
		output.distanceOffset.y = yList[index];
		output.distanceOffset.z = zList[index];


		return output;

	}
	__host__   inline void set(const int &index, const workItemGPU &input ){
		// Implement this
		this-> scales[index]= input.scale;
		this-> xList[index]= input.distanceOffset.x;
		this-> yList[index]= input.distanceOffset.y;
		this-> zList[index]= input.distanceOffset.z;

	}

	NEW_GPUQueue(unsigned long totalLength){
		this-> length = totalLength;
		this-> scales = new float[totalLength];
		this-> xList  = new float[totalLength];
		this-> yList  = new float[totalLength];
		this-> zList  = new float[totalLength];


	}
	NEW_GPUQueue(){
		// Invalid dummy-queue
		this-> length = 0;
		this-> scales = nullptr;
		this-> xList  = nullptr;
		this-> yList  = nullptr;
		this-> zList  = nullptr;
	}
	NEW_GPUQueue(const NEW_GPUQueue &blueprint){
		this-> length =   blueprint.length;
		this-> scales =   blueprint.scales;
		this-> xList  =  blueprint.xList;
		this-> yList  =  blueprint.yList;
		this-> zList  =  blueprint.zList;


	}



};










////////////////////////////////////////////////////////////////////



__global__ void renderMeshes(
        unsigned long totalItemsToRender,
        NEW_GPUQueue workQueue,
        GPUMesh* meshes,
        unsigned int meshCount,
        unsigned int width,
        unsigned int height,
        unsigned char* frameBuffer,
        int* depthBuffer
) {
	unsigned int item = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int triangleIndex = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int meshIndex = blockIdx.z;
	float4 v0,v1,v2;
	int isLarge = false;
	workItemGPU objectToRender;
	int Mask;
	if(item >= totalItemsToRender || meshIndex >= meshCount || triangleIndex >= meshes[meshIndex].vertexCount / 3) {

		// isLarge = false;

	}
	else{
    //for(unsigned int item = 0; item < totalItemsToRender; item++) {
	//for (unsigned int meshIndex = 0; meshIndex < meshCount; meshIndex++) {
    //for(unsigned int triangleIndex = 0; triangleIndex < meshes[meshIndex].vertexCount / 3; triangleIndex++) {

	objectToRender = workQueue.get(item);



	v0 = meshes[meshIndex].vertices[triangleIndex * 3 + 0];
	v1 = meshes[meshIndex].vertices[triangleIndex * 3 + 1];
	v2 = meshes[meshIndex].vertices[triangleIndex * 3 + 2];

	runVertexShader(v0, objectToRender.distanceOffset, objectToRender.scale, width, height);
	runVertexShader(v1, objectToRender.distanceOffset, objectToRender.scale, width, height);
	runVertexShader(v2, objectToRender.distanceOffset, objectToRender.scale, width, height);
	isLarge = f_isLarge(v0,v1,v2);
	Mask = __ballot_sync(0xFFFFFFFF, isLarge); // The mask is a queue.
	if( !isLarge ){ // Do the easy work
		rasteriseTriangle(v0, v1, v2, meshes[meshIndex], triangleIndex, frameBuffer, depthBuffer, width, height);
	}
	}


// Because everything else crashes, we now share the information form the
//difficult triangles, and
unsigned int PopC =__popc(Mask);
for(unsigned int i =0 ; i< PopC; ++i){
		//Everyone has the same triangle index, but in different items
		int first =  __ffs(Mask)-1;
		Mask -=(1<< (first)); // Deletes the bit

		// Share the most important data.
		int firstTrig= __shfl_sync(0xFFFFFFFF ,  triangleIndex, first);
		int firstItem= __shfl_sync(0xFFFFFFFF ,  item, first);
		int firstMesh = __shfl_sync(0xFFFFFFFF ,  meshIndex, first);


// Others do the work


	// I can not risk shuffeling, since it might break everything, but I'd
	// try to thansfer this information more elegantly (Preferably with a generic function)
	// So that I'd never have to look at it again.
																	// template <typename T>
																	// T share(T in, source){
																	//  T out;
																	//	int* ptrTo = (int*)&out;
																	//	int* ptrFrom = (int*)&in;
																	//	for(int i =0; i< (sizeof(T)+sizeof(int) -1)/sizeof(int) ; ++i){
																	//		*ptrOut = __shfl_sync(00xFFFFFFFF, *ptrIn, source);
																	//		++ptrOut;
																	//		++ptrin;
																	//		}
																	//	return out;
																	// }


// Now we do the exact same thing again, but with the shared variables instead
	workItemGPU objectToRender = workQueue.get(firstItem);


	// I suspect a slilent stack overflow or something. It we try to shuffle the vector everything crashes
	// We will insead abuse that we are acting on the same memory to read in a single cache-line

	// I want to run:
	// float4 Hv0 = getVec( v0, first);
	// float4 Hv1 = getVec( v1, first);
	// float4 Hv2 = getVec( v2, first);

	// But I must run: This causes extra latency from the memory
	float4 Hv0 = meshes[firstMesh].vertices[firstTrig * 3 + 0];
	float4 Hv1 = meshes[firstMesh].vertices[firstTrig * 3 + 1];
	float4 Hv2 = meshes[firstMesh].vertices[firstTrig * 3 + 2];


	runVertexShader(Hv0, objectToRender.distanceOffset, objectToRender.scale, width, height);
	runVertexShader(Hv1, objectToRender.distanceOffset, objectToRender.scale, width, height);
	runVertexShader(Hv2, objectToRender.distanceOffset, objectToRender.scale, width, height);

	// Because it works in this specific case, we put the functionality for
	// sharing large triangles inside the rasteriseTriangle, instead of outside it,
	// like reasonable people


	rasteriseTriangle(Hv0, Hv1, Hv2, meshes[firstMesh], firstTrig, frameBuffer, depthBuffer, width, height);



}

}



void fillWorkQueue(
        NEW_GPUQueue workQueue,
        float largestBoundingBoxSide,
        int depthLimit,
        unsigned long* nextIndexInQueue,
        float scale = 1.0,
        float3 distanceOffset = {0, 0, 0}) {

    // Queue a work item at the current scale and location
    workQueue.set(*nextIndexInQueue, {scale, distanceOffset});
    (*nextIndexInQueue)++;

    // Check whether we've reached the recursive depth of the fractal we want to reach
    depthLimit--;
    if(depthLimit == 0) {
        return;
    }

    // Now we recursively draw the meshes in a smaller size
    for(int offsetX = -1; offsetX <= 1; offsetX++) {
        for(int offsetY = -1; offsetY <= 1; offsetY++) {
            for(int offsetZ = -1; offsetZ <= 1; offsetZ++) {
                float3 offset = make_float3(offsetX,offsetY,offsetZ);
                // We draw the new objects in a grid around the "main" one.
                // We thus skip the location of the object itself.
                if(offsetX == 0 && offsetY == 0 && offsetZ == 0) {
                    continue;
                }

                float smallerScale = scale / 3.0f;
                float3 displacedOffset = make_float3(
                        distanceOffset.x + offset.x * (largestBoundingBoxSide / 2.0f) * scale,
                        distanceOffset.y + offset.y * (largestBoundingBoxSide / 2.0f) * scale,
                        distanceOffset.z + offset.z * (largestBoundingBoxSide / 2.0f) * scale
                );

                fillWorkQueue(workQueue, largestBoundingBoxSide, depthLimit, nextIndexInQueue, smallerScale, displacedOffset);
            }
        }
    }

}

__global__ void initialiseFramebuffer(unsigned char* frameBuffer, int width, int height) {
	unsigned int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

	if(threadIndex >= 4 * width * height) {
		return;
	}

		frameBuffer[threadIndex] = 255* (threadIndex % 4 == 3);

}

__global__ void initialiseDepthBuffer(int* depthBuffer, int width, int height) {
	unsigned int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadIndex >= width * height) {
		return;
	}

	depthBuffer[threadIndex] = 16777216;
}

// This function kicks off the rasterisation process.
std::vector<unsigned char> rasteriseGPU(std::string inputFile, unsigned int width, unsigned int height, unsigned int depthLimit) {
    std::cout << "Rendering an image on the GPU.." << std::endl;
    std::cout << "Loading '" << inputFile << "' file... " << std::endl;

    std::vector<GPUMesh> meshes = loadWavefrontGPU(inputFile, false);

    // We first need to allocate some buffers.
    // The framebuffer contains the image being rendered.
    unsigned char* frameBuffer = new unsigned char[width * height * 4];
    // The depth buffer is used to make sure that objects closer to the camera occlude/obscure objects that are behind it
    for (unsigned int i = 0; i < (4 * width * height); i+=4) {
		frameBuffer[i + 0] = 0;
		frameBuffer[i + 1] = 0;
		frameBuffer[i + 2] = 0;
		frameBuffer[i + 3] = 255;
	}

	int* depthBuffer = new int[width * height];
	for(unsigned int i = 0; i < width * height; i++) {
    	depthBuffer[i] = 1;
    }

    float3 boundingBoxMin = make_float3(std::numeric_limits<float>::max(), std::numeric_limits<float>::max(), std::numeric_limits<float>::max());
    float3 boundingBoxMax = make_float3(std::numeric_limits<float>::min(), std::numeric_limits<float>::min(), std::numeric_limits<float>::min());

    std::cout << "Rendering image... " << std::endl;

    for(unsigned int i = 0; i < meshes.size(); i++) {
        for(unsigned int vertex = 0; vertex < meshes.at(i).vertexCount; vertex++) {
            boundingBoxMin.x = std::min(boundingBoxMin.x, meshes.at(i).vertices[vertex].x);
            boundingBoxMin.y = std::min(boundingBoxMin.y, meshes.at(i).vertices[vertex].y);
            boundingBoxMin.z = std::min(boundingBoxMin.z, meshes.at(i).vertices[vertex].z);

            boundingBoxMax.x = std::max(boundingBoxMax.x, meshes.at(i).vertices[vertex].x);
            boundingBoxMax.y = std::max(boundingBoxMax.y, meshes.at(i).vertices[vertex].y);
            boundingBoxMax.z = std::max(boundingBoxMax.z, meshes.at(i).vertices[vertex].z);
        }
    }

    float3 boundingBoxDimensions = make_float3(
            boundingBoxMax.x - boundingBoxMin.x,
            boundingBoxMax.y - boundingBoxMin.y,
            boundingBoxMax.z - boundingBoxMin.z);
    float largestBoundingBoxSide = std::max(std::max(boundingBoxDimensions.x, boundingBoxDimensions.y), boundingBoxDimensions.z);


    unsigned char* device_frameBuffer;
    int* device_depthBuffer;

    checkCudaErrors(hipMalloc(&device_frameBuffer, width * height * 4 * sizeof(unsigned char)));
    checkCudaErrors(hipMalloc(&device_depthBuffer, width * height * sizeof(int)));

    const unsigned int initialisationBlockSize = 256;

    unsigned int blockCountFrameBuffer = ((width * height * 4) / initialisationBlockSize) + 1;
    initialiseFramebuffer<<<blockCountFrameBuffer, initialisationBlockSize>>>(device_frameBuffer, width, height);

    unsigned int blockCountDepthBuffer = ((width * height) / initialisationBlockSize) + 1;
    initialiseDepthBuffer<<<blockCountDepthBuffer, initialisationBlockSize>>>(device_depthBuffer, width, height);

    checkCudaErrors(hipDeviceSynchronize());



    // Each recursion level splits up the lowest level nodes into 28 smaller ones.
    // This regularity means we can calculate the total number of objects we need to render
    // which we can of course preallocate
    unsigned long totalItemsToRender = 0;
    for(unsigned long level = 0; level < depthLimit; level++) {
        totalItemsToRender += std::pow(26ul, level);
    }

		////////////////////////
		// Cnanged code
		NEW_GPUQueue workQueue(totalItemsToRender); //! (changed)
    // workItemGPU* workQueue = new workItemGPU[totalItemsToRender];

		////////////////////////

    std::cout << "Number of items to be rendered: " << totalItemsToRender << std::endl;

    unsigned long counter = 0;

    fillWorkQueue(workQueue, largestBoundingBoxSide, depthLimit, &counter);


		////////////////////////
		// Old code
		// unsigned long workQueueSizeBytes = totalItemsToRender * sizeof(workItemGPU);

    //workItemGPU* device_workQueue;


    // checkCudaErrors(hipMalloc(&device_workQueue, workQueueSizeBytes));
    // checkCudaErrors(hipMemcpy(device_workQueue, workQueue, workQueueSizeBytes, hipMemcpyHostToDevice));
		/////////////////////////////
		/////////////////////////////////////////////
		// NEW CODE
		NEW_GPUQueue device_workQueue; // No argumetns makes an empty dummy (changed)
		device_workQueue.length = totalItemsToRender;// Just in case
		int listSize = sizeof(float)* totalItemsToRender;// Same for whole queue
		checkCudaErrors(hipMalloc( &(device_workQueue.scales),  listSize));
		checkCudaErrors(hipMalloc( &(device_workQueue.xList) ,  listSize));
		checkCudaErrors(hipMalloc( &(device_workQueue.yList) ,  listSize));
		checkCudaErrors(hipMalloc( &(device_workQueue.zList) ,  listSize));


    checkCudaErrors(hipMemcpy(device_workQueue.scales, workQueue.scales, listSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_workQueue.xList , workQueue.xList , listSize, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(device_workQueue.yList , workQueue.yList , listSize, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(device_workQueue.zList , workQueue.zList , listSize, hipMemcpyHostToDevice));

		/////////////////////////////////////////////

	std::vector<GPUMesh> host_meshArray(meshes.begin(), meshes.end());
	for(int i = 0; i < meshes.size(); i++) {
		size_t vertexBufferSize = meshes.at(i).vertexCount * sizeof(float4);
		size_t normalBufferSize = meshes.at(i).vertexCount * sizeof(float3);

		checkCudaErrors(hipMalloc(&host_meshArray.at(i).vertices, vertexBufferSize));
		checkCudaErrors(hipMalloc(&host_meshArray.at(i).normals, normalBufferSize));

		checkCudaErrors(hipMemcpy(host_meshArray.at(i).vertices, meshes.at(i).vertices, vertexBufferSize, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(host_meshArray.at(i).normals, meshes.at(i).normals, normalBufferSize, hipMemcpyHostToDevice));
	}

	// Block x axis: Job queue
	// Block y axis: max vertex count
	// Block z axis: meshCount

	const unsigned int threadsPerWorkQueueBlock = 32;

	const unsigned int threadsPerVertexBlock = 3;
	GPUMesh* device_meshArray;
	checkCudaErrors(hipMalloc(&device_meshArray, meshes.size() * sizeof(GPUMesh)));
	checkCudaErrors(hipMemcpy(device_meshArray, host_meshArray.data(), meshes.size() * sizeof(GPUMesh), hipMemcpyHostToDevice));

	unsigned long maxMeshSize = 0;
	for(int i = 0; i < meshes.size(); i++) {
		maxMeshSize = std::max(maxMeshSize, meshes.at(i).vertexCount);
	}



	int jobQueueBlockCount = (totalItemsToRender / threadsPerWorkQueueBlock) + 1;
	int vertexBlockCount = (maxMeshSize / threadsPerVertexBlock) + 1;

	dim3 gridDimensions(  jobQueueBlockCount, vertexBlockCount, meshes.size());
	dim3 blockDimensions( threadsPerWorkQueueBlock, threadsPerVertexBlock, 1);
	timer.start();
	renderMeshes<<<gridDimensions, blockDimensions>>>(
		totalItemsToRender, device_workQueue,
		device_meshArray, meshes.size(),
		width, height, device_frameBuffer, device_depthBuffer);

	checkCudaErrors(hipDeviceSynchronize());
	timer.stop();
    std::cout << "Finished!" << std::endl;

    // Copy the output picture into a vector so that the image dump code is happy :)
    std::vector<unsigned char> outputFramebuffer(frameBuffer, frameBuffer + (width * height * 4));

   	checkCudaErrors(hipMemcpy(outputFramebuffer.data(), device_frameBuffer, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost));
   	hipDeviceReset();
		// TODO Free everything at the end
    return outputFramebuffer;
}
