#include "hip/hip_runtime.h"
#include "gpurasteriser.cuh"
#include "utilities/OBJLoader.hpp"
#include <vector>
#include <iomanip>
#include <chrono>
#include <limits>
#include <iostream>
#include <algorithm>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "utilities/cuda_error_helper.hpp"
#include "stdio.h"
#include ""
#include <chrono>

// UTILITY FUNCTIONS HAVE BEEN MOVED INTO THE KERNEL SOURCE FILE ITSELF
// CUDA relocatable and separable compilation is possible, but due to the many possible
// problems it can cause on different platforms, I decided to take the safe route instead
// and make sure it would compile fine for everyone. That implies moving everything into
// one file unfortunately.

class globalLight {
public:
	float3 direction;
	float3 colour;
	__device__ globalLight(float3 const vdirection, float3 const vcolour) : direction(vdirection), colour(vcolour) {}
};

__device__ float dotGPU(float3 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float3 normalizeGPU(float3 v)
{
    float invLen = 1.0f / sqrtf(dotGPU(v, v));
    v.x *= invLen;
    v.y *= invLen;
    v.z *= invLen;
    return v;
}

//////////////////////////////////////////////////////////////////
// New function declarations
struct workItemGPU;
__global__ void GPU_SetAlpha(unsigned char* frameBuffer, int arrayLength);
__global__ void GPU_fillDepthBuffer(int* GPU_depthBuffer, int arrayLength, int value);
GPUMesh* copyMeshesToGPU( std::vector<GPUMesh> &meshes);
__global__ void GPU_render(
				unsigned long totalItemsToRender,
				struct workItemGPU* workQueue,
				GPUMesh* meshes,
				unsigned int meshCount,
				unsigned int width,
				unsigned int height,
				unsigned char* frameBuffer,
				int* depthBuffer
);
//////////////////////////////////////////////////////////////////////





// Utility function if you'd like to convert the depth buffer to an integer format.
__device__ int depthFloatToInt(float value) {
	value = (value + 1.0f) * 0.5f;
    return static_cast<int>(static_cast<double>(value) * static_cast<double>(16777216));
}

__device__ bool isPointInTriangle(
		float4 const &v0, float4 const &v1, float4 const &v2,
		unsigned int const x, unsigned int const y,
		float &u, float &v, float &w) {
		u = (((v1.y - v2.y) * (x    - v2.x)) + ((v2.x - v1.x) * (y    - v2.y))) /
				 	 (((v1.y - v2.y) * (v0.x - v2.x)) + ((v2.x - v1.x) * (v0.y - v2.y)));
		if (u < 0) {
			return false;
		}
		v = (((v2.y - v0.y) * (x    - v2.x)) + ((v0.x - v2.x) * (y    - v2.y))) /
					(((v1.y - v2.y) * (v0.x - v2.x)) + ((v2.x - v1.x) * (v0.y - v2.y)));
		if (v < 0) {
			return false;
		}
		w = 1 - u - v;
		if (w < 0) {
			return false;
		}
		return true;
}

__device__ float3 computeInterpolatedNormal(
		float3 const &normal0,
		float3 const &normal1,
		float3 const &normal2,
		float3 const &weights
	) {
	float3 weightedN0, weightedN1, weightedN2;

	weightedN0.x = (normal0.x * weights.x);
	weightedN0.y = (normal0.y * weights.x);
	weightedN0.z = (normal0.z * weights.x);

	weightedN1.x = (normal1.x * weights.y);
	weightedN1.y = (normal1.y * weights.y);
	weightedN1.z = (normal1.z * weights.y);

	weightedN2.x = (normal2.x * weights.z);
	weightedN2.y = (normal2.y * weights.z);
	weightedN2.z = (normal2.z * weights.z);

	float3 weightedNormal;

	weightedNormal.x = weightedN0.x + weightedN1.x + weightedN2.x;
	weightedNormal.y = weightedN0.y + weightedN1.y + weightedN2.y;
	weightedNormal.z = weightedN0.z + weightedN1.z + weightedN2.z;

	return normalizeGPU(weightedNormal);
}

__device__ float computeDepth(
		float4 const &v0, float4 const &v1, float4 const &v2,
		float3 const &weights) {
	return weights.x * v0.z + weights.y * v1.z + weights.z * v2.z;
}





// ORIGINAL SOURCE FILE IS STARTING HERE

struct workItemGPU {
    float scale;
    float3 distanceOffset;

    workItemGPU(float& scale_, float3& distanceOffset_) : scale(scale_), distanceOffset(distanceOffset_) {}
    workItemGPU() : scale(1), distanceOffset(make_float3(0, 0, 0)) {}
};

// Added  __device__ (Jørn)
__device__ void runVertexShader( float4 &vertex,
                      float3 positionOffset,
                      float scale,
					  unsigned int const width,
					  unsigned int const height,
				  	  float const rotationAngle = 0)
{
	float const pi = 3.1415926f;
	// The matrices defined below are the ones used to transform the vertices and normals.

	// This projection matrix assumes a 16:9 aspect ratio, and an field of view (FOV) of 90 degrees.
	mat4x4 const projectionMatrix(
		0.347270,   0, 			0, 		0,
		0,	  		0.617370, 	0,		0,
		0,	  		0,			-1, 	-0.2f,
		0,	  		0,			-1,		0);

	mat4x4 translationMatrix(
		1,			0,			0,			0 + positionOffset.x /*X*/,
		0,			1,			0,			0 + positionOffset.y /*Y*/,
		0,			0,			1,			-10 + positionOffset.z /*Z*/,
		0,			0,			0,			1);

	mat4x4 scaleMatrix(
		scale/*X*/,	0,			0,				0,
		0, 			scale/*Y*/, 0,				0,
		0, 			0,			scale/*Z*/, 	0,
		0, 			0,			0,				1);

	mat4x4 const rotationMatrixX(
		1,			0,				0, 				0,
		0, 			cosf(0), 	-sinf(0),	0,
		0, 			sinf(0),	cosf(0), 	0,
		0, 			0,				0,				1);

	float const rotationAngleRad = (pi / 4.0f) + (rotationAngle / (180.0f/pi));

	mat4x4 const rotationMatrixY(
		cosf(rotationAngleRad), 0, sinf(rotationAngleRad), 0,
		0, 1, 0, 0,
		-sinf(rotationAngleRad), 0, cosf(rotationAngleRad), 	0,
		0, 0, 0, 1);

	mat4x4 const rotationMatrixZ(
		cosf(pi),	-sinf(pi),	0,			0,
		sinf(pi), 	cosf(pi), 	0,			0,
		0,				0,				1,			0,
		0, 				0,				0,			1);

	mat4x4 const MVP =
		projectionMatrix * translationMatrix * rotationMatrixX * rotationMatrixY * rotationMatrixZ * scaleMatrix;

		float4 transformed = (MVP * vertex);

    vertex.x = transformed.x / transformed.w;
    vertex.y = transformed.y / transformed.w;
    vertex.z = transformed.z / transformed.w;
    vertex.w = 1.0;

    vertex.x = (vertex.x + 0.5f) * (float) width;
    vertex.y = (vertex.y + 0.5f) * (float) height;
}

// Added  __device__ (Jørn)
__device__
void runFragmentShader( unsigned char* frameBuffer,
						unsigned int const baseIndex,
						GPUMesh &mesh,
						unsigned int triangleIndex,
						float3 const &weights)
{
	float3 normal = computeInterpolatedNormal(
            mesh.normals[3 * triangleIndex + 0],
            mesh.normals[3 * triangleIndex + 1],
            mesh.normals[3 * triangleIndex + 2],
			weights);

    float3 colour = make_float3(0.0f, 0.0f, 0.0f);

    const unsigned int lightSourceCount = 1;
    const globalLight lightSources[lightSourceCount] = {{make_float3(0.3f, 0.5f, 1.0f), make_float3(1.0f, 1.0f, 1.0f)}};

	for (unsigned int lightSource = 0; lightSource < lightSourceCount; lightSource++) {
		globalLight l = lightSources[lightSource];
		float lightNormalDotProduct =
			normal.x * l.direction.x + normal.y * l.direction.y + normal.z * l.direction.z;

		float3 diffuseReflectionColour;
		diffuseReflectionColour.x = mesh.objectDiffuseColour.x * l.colour.x;
		diffuseReflectionColour.y = mesh.objectDiffuseColour.y * l.colour.y;
		diffuseReflectionColour.z = mesh.objectDiffuseColour.z * l.colour.z;

		colour.x += diffuseReflectionColour.x * lightNormalDotProduct;
		colour.y += diffuseReflectionColour.y * lightNormalDotProduct;
		colour.z += diffuseReflectionColour.z * lightNormalDotProduct;
	}

    colour.x = fminf(fmaxf(colour.x, 0.0f), 1.0f);
    colour.y = fminf(fmaxf(colour.y, 0.0f), 1.0f);
    colour.z = fminf(fmaxf(colour.z, 0.0f), 1.0f);

    frameBuffer[4 * baseIndex + 0] = colour.x * 255.0f;
    frameBuffer[4 * baseIndex + 1] = colour.y * 255.0f;
    frameBuffer[4 * baseIndex + 2] = colour.z * 255.0f;
    frameBuffer[4 * baseIndex + 3] = 255;

}

/**
 * The main procedure which rasterises all triangles on the framebuffer
 * @param transformedMesh         Transformed mesh object
 * @param frameBuffer             frame buffer for the rendered image
 * @param depthBuffer             depth buffer for every pixel on the image
 * @param width                   width of the image
 * @param height                  height of the image
 */


 // Added __device__ (Jørn)
__device__ void rasteriseTriangle( float4 &v0, float4 &v1, float4 &v2,
                        GPUMesh &mesh,
                        unsigned int triangleIndex,
                        unsigned char* frameBuffer,
                        int* depthBuffer,
                        unsigned int const width,
                        unsigned int const height ) {

    // Compute the bounding box of the triangle.
    // Pixels that are intersecting with the triangle can only lie in this rectangle
	unsigned int minx = unsigned(floorf(fminf(fminf(v0.x, v1.x), v2.x)));
	unsigned int maxx = unsigned(ceilf(fmaxf(fmaxf(v0.x, v1.x), v2.x)));
	unsigned int miny = unsigned(floorf(fminf(fminf(v0.y, v1.y), v2.y)));
	unsigned int maxy = unsigned(ceilf(fmaxf(fmaxf(v0.y, v1.y), v2.y)));
	//std::cout<< "Probe: dx (maxx -minx)"<<maxx -minx<<"\n";

	// Make sure the screen coordinates stay inside the window
    // This ensures parts of the triangle that are outside the
    // view of the camera are not drawn.
	minx = fmaxf(minx, (unsigned int) 0);
	maxx = fminf(maxx, width);
	miny = fmaxf(miny, (unsigned int) 0);
	maxy = fminf(maxy, height);

	// We iterate over each pixel in the triangle's bounding box
	for (unsigned int x = minx; x < maxx; x++) {
		for (unsigned int y = miny; y < maxy; y++) {
			float u, v, w;
			// For each point in the bounding box, determine whether that point lies inside the triangle
			if (isPointInTriangle(v0, v1, v2, x, y, u, v, w)) {
				// If it does, compute the distance between that point on the triangle and the screen
				float pixelDepth = computeDepth(v0, v1, v2, make_float3(u, v, w));
				// If the point is closer than any point we have seen thus far, render it.
				// Otherwise it is hidden behind another object, and we can throw it away
				// Because it will be invisible anyway.
          if (pixelDepth >= -1 && pixelDepth <= 1) {
						int pixelDepthConverted = depthFloatToInt(pixelDepth);

					  // If it is, we update the depth buffer to the new depth.
						atomicMin(&(depthBuffer[y * width + x]), pixelDepthConverted);

           	if (pixelDepthConverted == depthBuffer[y * width + x]) {

							//depthBuffer[y * width + x] = pixelDepthConverted;

					    // And finally we determine the colour of the pixel, now that
					    // we know our pixel is the closest we have seen thus far.
							float3 temp;
							temp.x = u;
							temp.y = v;
							temp.z = w;
							// printf( "(u,v,w)=(%f,%f, %f)",temp.x, temp.y, temp.z);

							runFragmentShader(frameBuffer, x + (width * y), mesh, triangleIndex, temp);
						}
					}
				}
			}
		}
}

__device__
void renderMeshes(
        unsigned long totalItemsToRender,
        struct workItemGPU* workQueue,
        GPUMesh* meshes,
        unsigned int meshCount,
        unsigned int width,
        unsigned int height,
        unsigned char* frameBuffer,
        int* depthBuffer
) {
	//std::cout<< "Probe: meshCount "<<meshCount<<"\n"; // Probe
	//std::cout<< "Probe: meshes[0].vertexCount "<<meshes[0].vertexCount<<"\n"; // Probe


    for(unsigned int item = 0; item < totalItemsToRender; item++) {
        struct workItemGPU objectToRender = workQueue[item];
        for (unsigned int meshIndex = 0; meshIndex < meshCount; meshIndex++) {
            for(unsigned int triangleIndex = 0; triangleIndex < meshes[meshIndex].vertexCount / 3; triangleIndex++) {
                float4 v0 = meshes[meshIndex].vertices[triangleIndex * 3 + 0];
                float4 v1 = meshes[meshIndex].vertices[triangleIndex * 3 + 1];
                float4 v2 = meshes[meshIndex].vertices[triangleIndex * 3 + 2];

                runVertexShader(v0, objectToRender.distanceOffset, objectToRender.scale, width, height);
                runVertexShader(v1, objectToRender.distanceOffset, objectToRender.scale, width, height);
                runVertexShader(v2, objectToRender.distanceOffset, objectToRender.scale, width, height);

                rasteriseTriangle(v0, v1, v2, meshes[meshIndex], triangleIndex, frameBuffer, depthBuffer, width, height);
            }
        }
    }
}



void fillWorkQueue(
        workItemGPU* workQueue,
        float largestBoundingBoxSide,
        int depthLimit,
        unsigned long* nextIndexInQueue,
        float scale = 1.0,
        float3 distanceOffset = {0, 0, 0}) {

    // Queue a work item at the current scale and location
    workQueue[*nextIndexInQueue] = {scale, distanceOffset};
    (*nextIndexInQueue)++;

    // Check whether we've reached the recursive depth of the fractal we want to reach
    depthLimit--;
    if(depthLimit == 0) {
        return;
    }

    // Now we recursively draw the meshes in a smaller size
    for(int offsetX = -1; offsetX <= 1; offsetX++) {
        for(int offsetY = -1; offsetY <= 1; offsetY++) {
            for(int offsetZ = -1; offsetZ <= 1; offsetZ++) {
                float3 offset = make_float3(offsetX,offsetY,offsetZ);
                // We draw the new objects in a grid around the "main" one.
                // We thus skip the location of the object itself.
                if(offsetX == 0 && offsetY == 0 && offsetZ == 0) {
                    continue;
                }

                float smallerScale = scale / 3.0f;
                float3 displacedOffset = make_float3(
                        distanceOffset.x + offset.x * (largestBoundingBoxSide / 2.0f) * scale,
                        distanceOffset.y + offset.y * (largestBoundingBoxSide / 2.0f) * scale,
                        distanceOffset.z + offset.z * (largestBoundingBoxSide / 2.0f) * scale
                );

                fillWorkQueue(workQueue, largestBoundingBoxSide, depthLimit, nextIndexInQueue, smallerScale, displacedOffset);
            }
        }
    }

}

// This function kicks off the rasterisation process.
std::vector<unsigned char> rasteriseGPU(std::string inputFile, unsigned int width, unsigned int height, unsigned int depthLimit) {
		/////////////////////////////////////////////////////////////////////////////////////
		// Our code
		std::chrono::high_resolution_clock::time_point start;
		start = std::chrono::high_resolution_clock::now();
		int count;
		checkCudaErrors(
			hipGetDeviceCount(&count)
		);
		std::cout<< "Number of devices = "<< count<< "\n";
		hipDeviceProp_t prop;
		checkCudaErrors(
			hipGetDeviceProperties(&prop, 0)
		);
		std::cout<< "Device name "<< prop.name << "\n";

		std::cout<< "Warps size "<< prop.warpSize << "\n";

		std::cout<< "Max threads per block "<< prop.maxThreadsPerBlock << "\n";

		std::cout<< "Max threads dim "<< prop.maxThreadsDim[0] << "-" << prop.maxThreadsDim[1] << "-"<< prop.maxThreadsDim[2] << "\n";

		std::cout<< "Max grid size "<< prop.maxGridSize[0] << "-" << prop.maxGridSize[1] << "-"<< prop.maxGridSize[2] << "\n";

		checkCudaErrors(
			hipSetDevice(0)
		);


		/////////////////////////////////////////////////////////////////////////////////////
    std::cout << "Rendering an image on the GPU.." << std::endl;
    std::cout << "Loading '" << inputFile << "' file... " << std::endl;

    std::vector<GPUMesh> meshes = loadWavefrontGPU(inputFile, false);

    // We first need to allocate some buffers.
    // The framebuffer contains the image being rendered.
    unsigned char* frameBuffer = new unsigned char[width * height * 4];
    // The depth buffer is used to make sure that objects closer to the camera occlude/obscure objects that are behind it
    for (unsigned int i = 0; i < (4 * width * height); i+=4) {
		frameBuffer[i + 0] = 0;
		frameBuffer[i + 1] = 0;
		frameBuffer[i + 2] = 0;
		frameBuffer[i + 3] = 255;
		}

		int* depthBuffer = new int[width * height];
		for(unsigned int i = 0; i < width * height; i++) {
	    	depthBuffer[i] = 16777216; // = 2 ^ 24
    }
		/////////////////////////////////////////////////////////////////////////////////////
		// Our code
		unsigned char* GPU_frameBuffer;
		checkCudaErrors(hipMalloc(&GPU_frameBuffer,  (width * height * 4) * sizeof(frameBuffer[0])));
		checkCudaErrors(hipMemset(GPU_frameBuffer, 0,  (width * height) * sizeof(frameBuffer[0])));
		GPU_SetAlpha<<<1+((width*height)/32),32>>>((unsigned char*)GPU_frameBuffer, width*height*4);
		hipDeviceSynchronize();
		// GPU -> CPU
		checkCudaErrors(hipMemcpy( frameBuffer, GPU_frameBuffer, (width * height * 4) * sizeof(frameBuffer[0]), hipMemcpyDeviceToHost));
		std::cout << "Frame buffer size:"<<(width * height) * sizeof(frameBuffer[0])<< "\n";

		int* GPU_depthBuffer;
		checkCudaErrors(hipMalloc(&GPU_depthBuffer,  (width * height) * sizeof(depthBuffer[0])));
		GPU_fillDepthBuffer<<<(width*height)/32, 32>>>(GPU_depthBuffer, width*height, 16777216);
		hipDeviceSynchronize();
		// GPU -> CPU
		checkCudaErrors(hipMemcpy( depthBuffer, GPU_depthBuffer, (width * height) * sizeof(depthBuffer[0]), hipMemcpyDeviceToHost));
		for(int i =0; i< 10; ++i){
			std::cout << "db-	 "<<i<<": "<< (int)depthBuffer[i]<<"\n";
		}

//Copying meshes


		GPUMesh* GPU_meshes = copyMeshesToGPU(meshes);


		/////////////////////////////////////////////////////////////////////////////////////

    float3 boundingBoxMin = make_float3(std::numeric_limits<float>::max(), std::numeric_limits<float>::max(), std::numeric_limits<float>::max());
    float3 boundingBoxMax = make_float3(std::numeric_limits<float>::min(), std::numeric_limits<float>::min(), std::numeric_limits<float>::min());

    std::cout << "Rendering image... " << std::endl;

    for(unsigned int i = 0; i < meshes.size(); i++) {
        for(unsigned int vertex = 0; vertex < meshes.at(i).vertexCount; vertex++) {
            boundingBoxMin.x = std::min(boundingBoxMin.x, meshes.at(i).vertices[vertex].x);
            boundingBoxMin.y = std::min(boundingBoxMin.y, meshes.at(i).vertices[vertex].y);
            boundingBoxMin.z = std::min(boundingBoxMin.z, meshes.at(i).vertices[vertex].z);

            boundingBoxMax.x = std::max(boundingBoxMax.x, meshes.at(i).vertices[vertex].x);
            boundingBoxMax.y = std::max(boundingBoxMax.y, meshes.at(i).vertices[vertex].y);
            boundingBoxMax.z = std::max(boundingBoxMax.z, meshes.at(i).vertices[vertex].z);
        }
    }

    float3 boundingBoxDimensions = make_float3(
            boundingBoxMax.x - boundingBoxMin.x,
            boundingBoxMax.y - boundingBoxMin.y,
            boundingBoxMax.z - boundingBoxMin.z);
    float largestBoundingBoxSide = std::max(std::max(boundingBoxDimensions.x, boundingBoxDimensions.y), boundingBoxDimensions.z);

    // Each recursion level splits up the lowest level nodes into 28 smaller ones.
    // This regularity means we can calculate the total number of objects we need to render
    // which we can of course preallocate
    unsigned long totalItemsToRender = 0;
    for(unsigned long level = 0; level < depthLimit; level++) {
        totalItemsToRender += std::pow(26ul, level);
    }

    workItemGPU* workQueue = new workItemGPU[totalItemsToRender];
    std::cout << "Number of items to be rendered: " << totalItemsToRender << std::endl;
		//std::cout << "Probe: totalItemsToRender " << totalItemsToRender << std::endl; // Probe


    unsigned long counter = 0;
    fillWorkQueue(workQueue, largestBoundingBoxSide, depthLimit, &counter);

///////////////////////////////////////////////////////////////////////////
// OLD CODE
/*
// This part does the work (?)
	renderMeshes(
			totalItemsToRender, workQueue,
			meshes.data(), meshes.size(),
			width, height, frameBuffer, depthBuffer);

    std::cout << "Finished!" << std::endl;

    // Copy the output picture into a vector so that the image dump code is happy :)
    std::vector<unsigned char> outputFramebuffer(frameBuffer, frameBuffer + (width * height * 4));
		*/
/////////////////////////////////////////////////////////////////////////////////////////////////
//New code

// We need the work-queue on the GPU
workItemGPU* GPU_workQueue;
checkCudaErrors(hipMalloc(&GPU_workQueue,  totalItemsToRender* sizeof(workQueue[0])));
checkCudaErrors(hipMemcpy( GPU_workQueue, workQueue, totalItemsToRender* sizeof(workQueue[0]), hipMemcpyHostToDevice));

// We only need the frame-buffer back from the GPU

int maxTriangleCount=0;
for(int i =0; i< meshes.size(); ++i){
	if(meshes[i].vertexCount> maxTriangleCount)
		maxTriangleCount = meshes[i].vertexCount;
		//std::cout << "VC:"<< meshes[i].vertexCount<<"\n";
}
const unsigned int BLOCK_SIZE = 32;
printf("Tot items: %i, Meshes: %i, maxTriangleCount %i \n",totalItemsToRender, meshes.size(), (maxTriangleCount/3) );
dim3 loopDimensions(totalItemsToRender, meshes.size(), (maxTriangleCount/3+BLOCK_SIZE -1)/BLOCK_SIZE );

GPU_render<<<loopDimensions,BLOCK_SIZE>>>(
				totalItemsToRender,
				GPU_workQueue,
				GPU_meshes,
				meshes.size(),
				width,
				height,
				GPU_frameBuffer,
				GPU_depthBuffer
);
hipDeviceSynchronize();
printf("Rendered from GPU?\n");
// Divide into workers



checkCudaErrors(hipMemcpy(  frameBuffer, GPU_frameBuffer, (width * height * 4) * sizeof(frameBuffer[0]), hipMemcpyDeviceToHost));

std::vector<unsigned char> outputFramebuffer(frameBuffer, frameBuffer + (width * height * 4));
std::chrono::high_resolution_clock::time_point now;
now = std::chrono::high_resolution_clock::now();
std::chrono::high_resolution_clock::duration rastarizeDuration= now -start; 
std::cout << "Entire rasterization  ="<< 
std::chrono::duration_cast<std::chrono::milliseconds>(rastarizeDuration).count() <<" ms\n";
/////////////////////////////////////////////////////////////////////////////////////////////////


    return outputFramebuffer;
}

//////////////////////////////////////////////////////////////////
// New code, New functions


// GPU functions
__global__ void GPU_SetAlpha(unsigned char* GPU_frameBuffer, int arrayLength){
	if( (blockDim.x*blockIdx.x + threadIdx.x)*4 + 3< arrayLength){
		GPU_frameBuffer[(blockDim.x*blockIdx.x + threadIdx.x)*4 + 3] = 255;
	}
}


__global__ void GPU_fillDepthBuffer(int* GPU_depthBuffer, int arrayLength, int value){
	int ind = (blockDim.x*blockIdx.x + threadIdx.x);
	if(  ind < arrayLength){
		GPU_depthBuffer[ind] = value;
	}
}



__global__ void GPU_render(
				unsigned long totalItemsToRender,
				workItemGPU* workQueue,
				GPUMesh* meshes,
				unsigned int meshCount,
				unsigned int width,
				unsigned int height,
				unsigned char* GPU_frameBuffer,
				int* GPU_depthBuffer
) {
	unsigned int item, meshIndex, triangleIndex;
	item = blockIdx.x;
	meshIndex = blockIdx.y;
	triangleIndex = blockIdx.z*blockDim.x + threadIdx.x;
	//printf("work: %i, %i, %i \n", item, meshIndex,triangleIndex+ 1000000 );
	//printf("z_tot: (%i, %i, %i) \n",gridDim.x, gridDim.y, gridDim.z);

	if( item < totalItemsToRender
	   && meshIndex < meshCount
	   && triangleIndex < meshes[meshIndex].vertexCount/3) {



		workItemGPU objectToRender = workQueue[item];


		// Do stuff
		float4 v0 = meshes[meshIndex].vertices[triangleIndex * 3 + 0];
		float4 v1 = meshes[meshIndex].vertices[triangleIndex * 3 + 1];
		float4 v2 = meshes[meshIndex].vertices[triangleIndex * 3 + 2];

		//printf("Working item %i, meshInd %i, triInd %i\n", item, meshIndex, triangleIndex);

		//
		runVertexShader(v0, objectToRender.distanceOffset, objectToRender.scale, width, height);
		runVertexShader(v1, objectToRender.distanceOffset, objectToRender.scale, width, height);
		runVertexShader(v2, objectToRender.distanceOffset, objectToRender.scale, width, height);
		//

		// if( threadIdx.x ==0){
		// 	printf("x: %i, y:%i, z:%i, t:%i \n\t v0: (%f, %f, %f, %f)\t v0: (%f, %f, %f, %f)\n",
		// 	blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x,
		// 	v0.x,v0.y, v0.z ,v0.w,
		// 	meshes[meshIndex].vertices[triangleIndex * 3 + 0].x, meshes[meshIndex].vertices[triangleIndex * 3 + 0].y, meshes[meshIndex].vertices[triangleIndex * 3 + 0].z, meshes[meshIndex].vertices[triangleIndex * 3 + 0].w);
		// 	// printf("v0: (%f, %f, %f, %f)\t",  v0.x,v0.y, v0.z ,v0.w);
		// 	// printf("v0: (%f, %f, %f, %f)\n", v0.x,v0.y, v0.z ,v0.w );
		// }
		rasteriseTriangle(v0, v1, v2, meshes[meshIndex], triangleIndex, GPU_frameBuffer, GPU_depthBuffer, width, height);
	}
	// else {
	// 	printf("Item: %i/%lu, mesh %i/%i, triangle %i/%i\n", item, totalItemsToRender,
	// 		meshIndex , (int)meshCount,
	// 		 triangleIndex , (int)meshes[meshIndex].vertexCount/3);
	// }

}
// CPU functions
GPUMesh* copyMeshesToGPU( std::vector<GPUMesh> &meshes){
	std::vector<GPUMesh> CPU_meshes = std::vector<GPUMesh>(meshes.size());
	for(int i = 0; i < meshes.size(); i++){
		CPU_meshes[i] = meshes[i].clone();
		float4* GPU_vertices;
		checkCudaErrors(hipMalloc(&GPU_vertices, CPU_meshes[i].vertexCount * sizeof(CPU_meshes[i].vertices[0])));
		checkCudaErrors(hipMemcpy(GPU_vertices, CPU_meshes[i].vertices, CPU_meshes[i].vertexCount * sizeof(CPU_meshes[i].vertices[0]), hipMemcpyHostToDevice));
		delete[] CPU_meshes[i].vertices;
		CPU_meshes[i].vertices = GPU_vertices;

		float3* GPU_normals;
		checkCudaErrors(hipMalloc(&GPU_normals,  CPU_meshes[i].vertexCount * sizeof(CPU_meshes[i].normals[0])));
		checkCudaErrors(hipMemcpy(GPU_normals, CPU_meshes[i].normals, CPU_meshes[i].vertexCount * sizeof(CPU_meshes[i].normals[0]), hipMemcpyHostToDevice));
		delete[] CPU_meshes[i].normals;
		CPU_meshes[i].normals = GPU_normals;
	}
	GPUMesh* GPU_meshes;
	checkCudaErrors(hipMalloc(&GPU_meshes,  CPU_meshes.size() * sizeof(CPU_meshes[0])));
	checkCudaErrors(hipMemcpy(GPU_meshes, CPU_meshes.data(), CPU_meshes.size() * sizeof(CPU_meshes[0]), hipMemcpyHostToDevice));
	return GPU_meshes;
}
